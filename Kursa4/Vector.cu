#include "hip/hip_runtime.h"
#include "Vector.cuh"
#include "hip/hip_runtime.h"
#include ""

template <class T>
__global__ void addKernel(T *c, const T *a, const T *b, const unsigned int &N)
{
	unsigned int i = threadIdx.x + blockIdx.x*blockDim.x;
	if (i<N)
		c[i] = a[i] + b[i];
}
template <class T>
__global__ void mulKernel(const T* a, const T *b, T *c, const unsigned int &N)
{
	unsigned int i = threadIdx.x + blockIdx.x*blockDim.x;
	if (i<N)
		c[i] = a[i] * b[i];
}
template <class T, class X>
__global__ void mulByNum(const T*a, const X n, T* b, const unsigned int &N)
{
	unsigned int i = threadIdx.x + blockIdx.x*blockDim.x;
	if (i < N)
		b[i] = a[i] * n;
}

template <class T>
Vector<T>::Vector()
{
}

template <class T>
Vector<T>::~Vector()
{
}

template <class T>
Vector<T> Vector<T>::operator+(const Vector<T>& a, const Vector<T>& b)
{
	Vector<T> result;
	T* d_a;
	T* d_b;
	T* d_c;
	T* h_c;
	if (a.length() != b.length)
		return nullptr;
	unsigned int size = sizeof(T)*a.length();
	malloc(&h_c, size);
	hipMalloc(&d_a, size);
	hipMalloc(&d_b, size);
	hipMalloc(&d_c, size);
	hipMemcpy(d_a, &a[0], size, hipMemcpyHostToDevice); 
	hipMemcpy(d_b, &b[0], size, hipMemcpyHostToDevice);
	unsigned int threadsperblock = 256;
	unsigned int blockspergrid = (a.length() + threadsperblock - 1) / threadsperblock;
	addKernel << <blockspergrid, threadsperblock >> > (d_a, d_b, d_c, size);
	hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);
	result.assign(h_c, h_c + a.length());
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	free(h_c);
	return result;
}

template <class T>
float Vector<T>::operator*(const Vector<T>&, const Vector<T>&)
{
}

template <class T>
Vector<T> Vector<T>::cross_multiple(const Vector<T>&)
{
}

template <class T>
Vector<T> Vector<T>::mixed_multiple(const Vector<T>&)
{
}

template <class T, class X>
Vector<T> operator *(const Vector<T>& a, const X& b)
{
	Vector<T> result;
	T* d_a;
	T* d_b;
	T* h_c;
	T d_x;
	hipMalloc(&d_x, sizeof(T));
	cudaMemCpy(d_x, b, sizeof(T), hipMemcpyHostToDevice);
	unsigned int size = sizeof(T)*a.length();
	malloc(&h_c, size);
	hipMalloc(&d_a, size);
	hipMalloc(&d_b, size);
	hipMemcpy(d_a, &a[0], size, hipMemcpyHostToDevice);
	unsigned int threadsperblock = 256;
	unsigned int blockspergrid = (a.length() + threadsperblock - 1) / threadsperblock;
	mulByNum<<<blockspergrid, threadsperblock >>> (d_a, d_x, size);
	hipMemcpy(h_c, d_b, size, hipMemcpyDeviceToHost);
	result.assign(h_c, h_c + a.length());
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_x);
	free(h_c);
	return result;
}

/*hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	hipError_t cudaStatus;
	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	addKernel << <1, size >> >(dev_c, dev_a, dev_b);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}*/